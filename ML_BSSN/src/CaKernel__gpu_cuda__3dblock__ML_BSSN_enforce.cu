#include"CaKernel__gpu_cuda__3dblock__ML_BSSN_enforce.h"
#include"CaKernel__ML_BSSN_enforce.code"
#include"CaKernel__gpu_cuda__mem.h"



                                       
  

                                       
  #undef MinimumLapse 



#include<assert.h>

void CAKERNEL_Launch_ML_BSSN_enforce(CCTK_ARGUMENTS)
{
    DECLARE_CCTK_ARGUMENTS;
    DECLARE_CCTK_PARAMETERS;

#define TS_ERROR(d)                                                            \
    if ( CAKERNEL_Tile##d <= stncl_##d##n + stncl_##d##p )                     \
      CCTK_VWarn                                                               \
       (CCTK_WARN_ABORT, __LINE__,__FILE__, CCTK_THORNSTRING,                  \
        "Tile size for ML_BSSN_enforce along %s axis (%d) too small for stencil %d -- %d\n",\
        #d, CAKERNEL_Tile##d, stncl_##d##n, stncl_##d##p);

    TS_ERROR(x); TS_ERROR(y); TS_ERROR(z);
#   undef TS_ERROR

    int vi = 0;
    
    
      assert((vi = CCTK_VarIndex("ML_BSSN::alpha"))>=0);
      void * d_alpha = Device_GetVarI(cctkGH, vi, 0); 
      void * d_alpha_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At11"))>=0);
      void * d_At11 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At11_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At12"))>=0);
      void * d_At12 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At12_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At13"))>=0);
      void * d_At13 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At13_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At22"))>=0);
      void * d_At22 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At22_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At23"))>=0);
      void * d_At23 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At23_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At33"))>=0);
      void * d_At33 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At33_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt11"))>=0);
      void * d_gt11 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt11_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt12"))>=0);
      void * d_gt12 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt12_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt13"))>=0);
      void * d_gt13 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt13_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt22"))>=0);
      void * d_gt22 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt22_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt23"))>=0);
      void * d_gt23 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt23_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt33"))>=0);
      void * d_gt33 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt33_out = Device_GetVarI(cctkGH, vi, 1);
    
    
    size_t datasize = cctk_lsh[0] * cctk_lsh[1] * cctk_lsh[2];

    
    
    const int blocky = iDivUp(cctk_lsh[1] - stncl_yn - stncl_yp, CAKERNEL_Tiley - stncl_yn - stncl_yp);

    CaCUDA_Kernel_Launch_Parameters prms(cctk_iteration,
        cctk_lsh[0], cctk_lsh[1], cctk_lsh[2],
        cctk_nghostzones[0], cctk_nghostzones[1], cctk_nghostzones[2], blocky,
        cctk_delta_space[0], cctk_delta_space[1], cctk_delta_space[2],
        cctk_delta_time,
        cctk_origin_space[0], cctk_origin_space[1], cctk_origin_space[2],
        cctk_time);


    CAKERNEL_ML_BSSN_enforce<<<                                                    
 dim3(iDivUp(prms.cagh_ni - stncl_xn - stncl_xp, CAKERNEL_Tilex - stncl_xn - stncl_xp), 
      iDivUp(prms.cagh_nk - stncl_zn - stncl_zp, CAKERNEL_Tilez - stncl_zn - stncl_zp) 
          * blocky),
 dim3(CAKERNEL_Tilex, CAKERNEL_Threadsy, CAKERNEL_Threadsz)>>>(
      

(typeof(alpha)) d_alpha,(typeof(At11)) d_At11,(typeof(At12)) d_At12,(typeof(At13)) d_At13,(typeof(At22)) d_At22,(typeof(At23)) d_At23,(typeof(At33)) d_At33,(typeof(gt11)) d_gt11,(typeof(gt12)) d_gt12,(typeof(gt13)) d_gt13,(typeof(gt22)) d_gt22,(typeof(gt23)) d_gt23,(typeof(gt33)) d_gt33,



 d_pars.ptr, 
d_pars.MinimumLapse_offset, 
 prms);
//    cutilCheckMsg("failed while updating the velocity");
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    
    
}
