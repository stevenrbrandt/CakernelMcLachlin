#include"CaKernel__gpu_cuda__3dblock__ML_BSSN_RHS2.h"
#include"CaKernel__ML_BSSN_RHS2.code"
#include"CaKernel__gpu_cuda__mem.h"



                                       
  

                                       
  #undef conformalMethod 



#include<assert.h>

void CAKERNEL_Launch_ML_BSSN_RHS2(CCTK_ARGUMENTS)
{
    DECLARE_CCTK_ARGUMENTS;
    DECLARE_CCTK_PARAMETERS;

#define TS_ERROR(d)                                                            \
    if ( CAKERNEL_Tile##d <= stncl_##d##n + stncl_##d##p )                     \
      CCTK_VWarn                                                               \
       (CCTK_WARN_ABORT, __LINE__,__FILE__, CCTK_THORNSTRING,                  \
        "Tile size for ML_BSSN_RHS2 along %s axis (%d) too small for stencil %d -- %d\n",\
        #d, CAKERNEL_Tile##d, stncl_##d##n, stncl_##d##p);

    TS_ERROR(x); TS_ERROR(y); TS_ERROR(z);
#   undef TS_ERROR

    int vi = 0;
    
    
      assert((vi = CCTK_VarIndex("ML_BSSN::alpha"))>=0);
      void * d_alpha = Device_GetVarI(cctkGH, vi, 0); 
      void * d_alpha_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At11"))>=0);
      void * d_At11 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At11_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At11rhs"))>=0);
      void * d_At11rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At11rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At12"))>=0);
      void * d_At12 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At12_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At12rhs"))>=0);
      void * d_At12rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At12rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At13"))>=0);
      void * d_At13 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At13_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At13rhs"))>=0);
      void * d_At13rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At13rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At22"))>=0);
      void * d_At22 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At22_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At22rhs"))>=0);
      void * d_At22rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At22rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At23"))>=0);
      void * d_At23 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At23_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At23rhs"))>=0);
      void * d_At23rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At23rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At33"))>=0);
      void * d_At33 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At33_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At33rhs"))>=0);
      void * d_At33rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At33rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::beta1"))>=0);
      void * d_beta1 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_beta1_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::beta2"))>=0);
      void * d_beta2 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_beta2_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::beta3"))>=0);
      void * d_beta3 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_beta3_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt11"))>=0);
      void * d_gt11 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt11_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt12"))>=0);
      void * d_gt12 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt12_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt13"))>=0);
      void * d_gt13 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt13_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt22"))>=0);
      void * d_gt22 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt22_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt23"))>=0);
      void * d_gt23 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt23_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt33"))>=0);
      void * d_gt33 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt33_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::phi"))>=0);
      void * d_phi = Device_GetVarI(cctkGH, vi, 0); 
      void * d_phi_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::trK"))>=0);
      void * d_trK = Device_GetVarI(cctkGH, vi, 0); 
      void * d_trK_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::Xt1"))>=0);
      void * d_Xt1 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_Xt1_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::Xt2"))>=0);
      void * d_Xt2 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_Xt2_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::Xt3"))>=0);
      void * d_Xt3 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_Xt3_out = Device_GetVarI(cctkGH, vi, 1);
    
    
    size_t datasize = cctk_lsh[0] * cctk_lsh[1] * cctk_lsh[2];

    
    
    const int blocky = iDivUp(cctk_lsh[1] - stncl_yn - stncl_yp, CAKERNEL_Tiley - stncl_yn - stncl_yp);

    CaCUDA_Kernel_Launch_Parameters prms(cctk_iteration,
        cctk_lsh[0], cctk_lsh[1], cctk_lsh[2],
        cctk_nghostzones[0], cctk_nghostzones[1], cctk_nghostzones[2], blocky,
        cctk_delta_space[0], cctk_delta_space[1], cctk_delta_space[2],
        cctk_delta_time,
        cctk_origin_space[0], cctk_origin_space[1], cctk_origin_space[2],
        cctk_time);


    CAKERNEL_ML_BSSN_RHS2<<<                                                    
 dim3(iDivUp(prms.cagh_ni - stncl_xn - stncl_xp, CAKERNEL_Tilex - stncl_xn - stncl_xp), 
      iDivUp(prms.cagh_nk - stncl_zn - stncl_zp, CAKERNEL_Tilez - stncl_zn - stncl_zp) 
          * blocky),
 dim3(CAKERNEL_Tilex, CAKERNEL_Threadsy, CAKERNEL_Threadsz)>>>(
      

(typeof(alpha)) d_alpha,(typeof(At11)) d_At11,(typeof(At11rhs)) d_At11rhs,(typeof(At12)) d_At12,(typeof(At12rhs)) d_At12rhs,(typeof(At13)) d_At13,(typeof(At13rhs)) d_At13rhs,(typeof(At22)) d_At22,(typeof(At22rhs)) d_At22rhs,(typeof(At23)) d_At23,(typeof(At23rhs)) d_At23rhs,(typeof(At33)) d_At33,(typeof(At33rhs)) d_At33rhs,(typeof(beta1)) d_beta1,(typeof(beta2)) d_beta2,(typeof(beta3)) d_beta3,(typeof(gt11)) d_gt11,(typeof(gt12)) d_gt12,(typeof(gt13)) d_gt13,(typeof(gt22)) d_gt22,(typeof(gt23)) d_gt23,(typeof(gt33)) d_gt33,(typeof(phi)) d_phi,(typeof(trK)) d_trK,(typeof(Xt1)) d_Xt1,(typeof(Xt2)) d_Xt2,(typeof(Xt3)) d_Xt3,



 d_pars.ptr, 
d_pars.conformalMethod_offset, 
 prms);
//    cutilCheckMsg("failed while updating the velocity");
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    
    
}
