#include"CaKernel__gpu_cuda__shared.h"
#include"CaKernel__gpu_cuda__mem.h"
/* Assume Piraha will generate this file and this file will be pushed here as well */

#include "cctk.h"
#include "cctk_Parameters.h"
#include "cctk_Arguments.h"

#ifdef __HIPCC__ 
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h> 
#endif

#include<assert.h>
#include<algorithm>
using namespace std;

 



Vars d_vars;
Pars d_pars;

void CaKernel_AllocDevVars(CCTK_ARGUMENTS)
{
  DECLARE_CCTK_ARGUMENTS;
  DECLARE_CCTK_PARAMETERS;

  
}

void CaKernel_FreeDevVars(){
  
}

void CaKernel_AllocDevPars(CCTK_ARGUMENTS)
{
  
  DECLARE_CCTK_ARGUMENTS;
  DECLARE_CCTK_PARAMETERS;
  
  int numvars =  13 ;
  size_t sizes[ 13  + 1] = {
    
      sizeof (CCTK_REAL), 
      sizeof (CCTK_REAL), 
      sizeof (CCTK_REAL), 
      sizeof (CCTK_REAL), 
      sizeof (CCTK_REAL), 
      sizeof (CCTK_INT), 
      sizeof (CCTK_REAL), 
      sizeof (CCTK_INT), 
      sizeof (CCTK_REAL), 
      sizeof (CCTK_REAL), 
      sizeof (CCTK_REAL), 
      sizeof (CCTK_INT), 
      sizeof (CCTK_REAL), 
      16 // to make sure that the "structure" is aligned to 16 bytes
    };   

  size_t elements[ 13  + 1] = {
      1,   1,   1,   1,   1,   1,   1,   1,   1,   1,   1,   1,   1, 
      0
    };   

  int offsets[ 13  + 1];   
  
  offsets[0] = 0;
  for (int i = 1; i <= numvars; i++){
    int oldoffset = offsets[i - 1] + sizes[i - 1] * elements[i - 1];
    for(int b = 2; b <= 16 && b <= sizes[i]; b <<= 1)
      oldoffset += (b - (oldoffset % b)) % b;
    offsets[i] =  oldoffset;
  }  
  
   
    d_pars.MinimumLapse_offset = offsets[(int)0];  
    d_pars.LapseAdvectionCoeff_offset = offsets[(int)1.0];  
    d_pars.BetaDriver_offset = offsets[(int)2.0];  
    d_pars.EpsDiss_offset = offsets[(int)3.0];  
    d_pars.ShiftBCoeff_offset = offsets[(int)4.0];  
    d_pars.harmonicN_offset = offsets[(int)5.0];  
    d_pars.ShiftAdvectionCoeff_offset = offsets[(int)6.0];  
    d_pars.harmonicShift_offset = offsets[(int)7.0];  
    d_pars.harmonicF_offset = offsets[(int)8.0];  
    d_pars.ShiftGammaCoeff_offset = offsets[(int)9.0];  
    d_pars.LapseACoeff_offset = offsets[(int)10.0];  
    d_pars.conformalMethod_offset = offsets[(int)11.0];  
    d_pars.AlphaDriver_offset = offsets[(int)12.0]; 

//  d_pars = pars;
  char * tmpptr = (char *)malloc (offsets[numvars]);
  CUDA_SAFE_CALL (hipMalloc ((void **) &(d_pars.ptr), offsets[numvars]));
  
   
  { 
    CCTK_REAL * ptr = (CCTK_REAL *) &tmpptr[d_pars.MinimumLapse_offset];
    *ptr = MinimumLapse;    
  }  
  { 
    CCTK_REAL * ptr = (CCTK_REAL *) &tmpptr[d_pars.LapseAdvectionCoeff_offset];
    *ptr = LapseAdvectionCoeff;    
  }  
  { 
    CCTK_REAL * ptr = (CCTK_REAL *) &tmpptr[d_pars.BetaDriver_offset];
    *ptr = BetaDriver;    
  }  
  { 
    CCTK_REAL * ptr = (CCTK_REAL *) &tmpptr[d_pars.EpsDiss_offset];
    *ptr = EpsDiss;    
  }  
  { 
    CCTK_REAL * ptr = (CCTK_REAL *) &tmpptr[d_pars.ShiftBCoeff_offset];
    *ptr = ShiftBCoeff;    
  }  
  { 
    CCTK_INT * ptr = (CCTK_INT *) &tmpptr[d_pars.harmonicN_offset];
    *ptr = harmonicN;    
  }  
  { 
    CCTK_REAL * ptr = (CCTK_REAL *) &tmpptr[d_pars.ShiftAdvectionCoeff_offset];
    *ptr = ShiftAdvectionCoeff;    
  }  
  { 
    CCTK_INT * ptr = (CCTK_INT *) &tmpptr[d_pars.harmonicShift_offset];
    *ptr = harmonicShift;    
  }  
  { 
    CCTK_REAL * ptr = (CCTK_REAL *) &tmpptr[d_pars.harmonicF_offset];
    *ptr = harmonicF;    
  }  
  { 
    CCTK_REAL * ptr = (CCTK_REAL *) &tmpptr[d_pars.ShiftGammaCoeff_offset];
    *ptr = ShiftGammaCoeff;    
  }  
  { 
    CCTK_REAL * ptr = (CCTK_REAL *) &tmpptr[d_pars.LapseACoeff_offset];
    *ptr = LapseACoeff;    
  }  
  { 
    CCTK_INT * ptr = (CCTK_INT *) &tmpptr[d_pars.conformalMethod_offset];
    *ptr = conformalMethod;    
  }  
  { 
    CCTK_REAL * ptr = (CCTK_REAL *) &tmpptr[d_pars.AlphaDriver_offset];
    *ptr = AlphaDriver;    
  } 
  
  CUDA_SAFE_CALL (hipMemcpy(d_pars.ptr, tmpptr, offsets[numvars], hipMemcpyHostToDevice));
  free(tmpptr);
  
}

void CaKernel_FreeDevPars()
{
  
//  free(pars.ptr);
  CUDA_SAFE_CALL(hipFree ((void *) d_pars.ptr));
  
}

void CaKernel_AllocDevMem (CCTK_ARGUMENTS)
{
  DECLARE_CCTK_ARGUMENTS;
  DECLARE_CCTK_PARAMETERS;
  int num_tl = 1;
  int vi = -1;
  
    assert((vi = CCTK_VarIndex("ML_BSSN::A"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::alpha"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::alpharhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::Arhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::At11"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::At11rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::At12"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::At12rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::At13"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::At13rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::At22"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::At22rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::At23"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::At23rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::At33"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::At33rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::B1"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::B1rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::B2"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::B2rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::B3"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::B3rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::beta1"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::beta1rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::beta2"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::beta2rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::beta3"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::beta3rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::gt11"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::gt11rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::gt12"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::gt12rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::gt13"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::gt13rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::gt22"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::gt22rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::gt23"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::gt23rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::gt33"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::gt33rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::phi"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::phirhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::trK"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::trKrhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::Xt1"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::Xt1rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::Xt2"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::Xt2rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::Xt3"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  
    assert((vi = CCTK_VarIndex("ML_BSSN::Xt3rhs"))>=0);
    num_tl = CCTK_MaxTimeLevelsVI(vi);
    Device_RegisterMem(CCTK_PASS_CTOC, vi, num_tl); 
  

  
  CaKernel_AllocDevPars(CCTK_PASS_CTOC);
  CaKernel_AllocDevVars(CCTK_PASS_CTOC);
}

/**
 * Free the memory of grid variables on GPU devices.
 */

int CaKernel_FreeDevMem () {
  
  CaKernel_FreeDevPars();
  CaKernel_FreeDevVars();
}


void CaKernel_InitDevice(CCTK_ARGUMENTS){
  DECLARE_CCTK_ARGUMENTS;
  DECLARE_CCTK_PARAMETERS;
  
  int num_dev; 
  CUDA_SAFE_CALL (hipGetDeviceCount (&num_dev));
  if (num_dev == 0)
  {
    CCTK_WARN
      (0, "There are no CUDA devices available. (They may be busy, or the driver may not be installed.)");
    exit(-1);
  }

  int myproc, devidx;

  myproc = CCTK_MyProc(cctkGH);
  devidx = myproc % num_dev;

  /* we set device based on the number of devices available on each node */
  CUDA_SAFE_CALL (hipSetDevice (devidx));
  CCTK_VInfo(CCTK_THORNSTRING, "number of device %d", num_dev);
  CCTK_VInfo(CCTK_THORNSTRING, "device %d is successfully assigned to process %d", devidx, myproc);
}



