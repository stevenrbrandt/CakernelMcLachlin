#include"CaKernel__gpu_cuda__3dblock__ML_BSSN_RHSStaticBoundary.h"
#include"CaKernel__ML_BSSN_RHSStaticBoundary.code"
#include"CaKernel__gpu_cuda__mem.h"



                                       
  

                                       
  


#include<assert.h>

void CAKERNEL_Launch_ML_BSSN_RHSStaticBoundary(CCTK_ARGUMENTS)
{
    DECLARE_CCTK_ARGUMENTS;
    DECLARE_CCTK_PARAMETERS;

#define TS_ERROR(d)                                                            \
    if ( CAKERNEL_Tile##d <= stncl_##d##n + stncl_##d##p )                     \
      CCTK_VWarn                                                               \
       (CCTK_WARN_ABORT, __LINE__,__FILE__, CCTK_THORNSTRING,                  \
        "Tile size for ML_BSSN_RHSStaticBoundary along %s axis (%d) too small for stencil %d -- %d\n",\
        #d, CAKERNEL_Tile##d, stncl_##d##n, stncl_##d##p);

    TS_ERROR(x); TS_ERROR(y); TS_ERROR(z);
#   undef TS_ERROR

    int vi = 0;
    
    
      assert((vi = CCTK_VarIndex("ML_BSSN::alpharhs"))>=0);
      void * d_alpharhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_alpharhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::Arhs"))>=0);
      void * d_Arhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_Arhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At11rhs"))>=0);
      void * d_At11rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At11rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At12rhs"))>=0);
      void * d_At12rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At12rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At13rhs"))>=0);
      void * d_At13rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At13rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At22rhs"))>=0);
      void * d_At22rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At22rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At23rhs"))>=0);
      void * d_At23rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At23rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At33rhs"))>=0);
      void * d_At33rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At33rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::B1rhs"))>=0);
      void * d_B1rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_B1rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::B2rhs"))>=0);
      void * d_B2rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_B2rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::B3rhs"))>=0);
      void * d_B3rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_B3rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::beta1rhs"))>=0);
      void * d_beta1rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_beta1rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::beta2rhs"))>=0);
      void * d_beta2rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_beta2rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::beta3rhs"))>=0);
      void * d_beta3rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_beta3rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt11rhs"))>=0);
      void * d_gt11rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt11rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt12rhs"))>=0);
      void * d_gt12rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt12rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt13rhs"))>=0);
      void * d_gt13rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt13rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt22rhs"))>=0);
      void * d_gt22rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt22rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt23rhs"))>=0);
      void * d_gt23rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt23rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt33rhs"))>=0);
      void * d_gt33rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt33rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::phirhs"))>=0);
      void * d_phirhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_phirhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::trKrhs"))>=0);
      void * d_trKrhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_trKrhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::Xt1rhs"))>=0);
      void * d_Xt1rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_Xt1rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::Xt2rhs"))>=0);
      void * d_Xt2rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_Xt2rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::Xt3rhs"))>=0);
      void * d_Xt3rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_Xt3rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
    
    size_t datasize = cctk_lsh[0] * cctk_lsh[1] * cctk_lsh[2];

    
    
    const int blocky = iDivUp(cctk_lsh[1] - stncl_yn - stncl_yp, CAKERNEL_Tiley - stncl_yn - stncl_yp);

    CaCUDA_Kernel_Launch_Parameters prms(cctk_iteration,
        cctk_lsh[0], cctk_lsh[1], cctk_lsh[2],
        cctk_nghostzones[0], cctk_nghostzones[1], cctk_nghostzones[2], blocky,
        cctk_delta_space[0], cctk_delta_space[1], cctk_delta_space[2],
        cctk_delta_time,
        cctk_origin_space[0], cctk_origin_space[1], cctk_origin_space[2],
        cctk_time);


    CAKERNEL_ML_BSSN_RHSStaticBoundary<<<                                                    
 dim3(iDivUp(prms.cagh_ni - stncl_xn - stncl_xp, CAKERNEL_Tilex - stncl_xn - stncl_xp), 
      iDivUp(prms.cagh_nk - stncl_zn - stncl_zp, CAKERNEL_Tilez - stncl_zn - stncl_zp) 
          * blocky),
 dim3(CAKERNEL_Tilex, CAKERNEL_Threadsy, CAKERNEL_Threadsz)>>>(
      

(typeof(alpharhs)) d_alpharhs,(typeof(Arhs)) d_Arhs,(typeof(At11rhs)) d_At11rhs,(typeof(At12rhs)) d_At12rhs,(typeof(At13rhs)) d_At13rhs,(typeof(At22rhs)) d_At22rhs,(typeof(At23rhs)) d_At23rhs,(typeof(At33rhs)) d_At33rhs,(typeof(B1rhs)) d_B1rhs,(typeof(B2rhs)) d_B2rhs,(typeof(B3rhs)) d_B3rhs,(typeof(beta1rhs)) d_beta1rhs,(typeof(beta2rhs)) d_beta2rhs,(typeof(beta3rhs)) d_beta3rhs,(typeof(gt11rhs)) d_gt11rhs,(typeof(gt12rhs)) d_gt12rhs,(typeof(gt13rhs)) d_gt13rhs,(typeof(gt22rhs)) d_gt22rhs,(typeof(gt23rhs)) d_gt23rhs,(typeof(gt33rhs)) d_gt33rhs,(typeof(phirhs)) d_phirhs,(typeof(trKrhs)) d_trKrhs,(typeof(Xt1rhs)) d_Xt1rhs,(typeof(Xt2rhs)) d_Xt2rhs,(typeof(Xt3rhs)) d_Xt3rhs,





 prms);
//    cutilCheckMsg("failed while updating the velocity");
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    
    
}
