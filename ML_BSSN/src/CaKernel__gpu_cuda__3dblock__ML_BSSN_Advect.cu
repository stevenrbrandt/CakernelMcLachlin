#include"CaKernel__gpu_cuda__3dblock__ML_BSSN_Advect.h"
#include"CaKernel__ML_BSSN_Advect.code"
#include"CaKernel__gpu_cuda__mem.h"



                                       
  

                                       
  #undef LapseAdvectionCoeff 
#undef ShiftBCoeff 
#undef LapseACoeff 
#undef ShiftAdvectionCoeff 



#include<assert.h>

void CAKERNEL_Launch_ML_BSSN_Advect(CCTK_ARGUMENTS)
{
    DECLARE_CCTK_ARGUMENTS;
    DECLARE_CCTK_PARAMETERS;

#define TS_ERROR(d)                                                            \
    if ( CAKERNEL_Tile##d <= stncl_##d##n + stncl_##d##p )                     \
      CCTK_VWarn                                                               \
       (CCTK_WARN_ABORT, __LINE__,__FILE__, CCTK_THORNSTRING,                  \
        "Tile size for ML_BSSN_Advect along %s axis (%d) too small for stencil %d -- %d\n",\
        #d, CAKERNEL_Tile##d, stncl_##d##n, stncl_##d##p);

    TS_ERROR(x); TS_ERROR(y); TS_ERROR(z);
#   undef TS_ERROR

    int vi = 0;
    
    
      assert((vi = CCTK_VarIndex("ML_BSSN::A"))>=0);
      void * d_A = Device_GetVarI(cctkGH, vi, 0); 
      void * d_A_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::alpha"))>=0);
      void * d_alpha = Device_GetVarI(cctkGH, vi, 0); 
      void * d_alpha_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::alpharhs"))>=0);
      void * d_alpharhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_alpharhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::Arhs"))>=0);
      void * d_Arhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_Arhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At11"))>=0);
      void * d_At11 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At11_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At11rhs"))>=0);
      void * d_At11rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At11rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At12"))>=0);
      void * d_At12 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At12_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At12rhs"))>=0);
      void * d_At12rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At12rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At13"))>=0);
      void * d_At13 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At13_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At13rhs"))>=0);
      void * d_At13rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At13rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At22"))>=0);
      void * d_At22 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At22_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At22rhs"))>=0);
      void * d_At22rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At22rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At23"))>=0);
      void * d_At23 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At23_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At23rhs"))>=0);
      void * d_At23rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At23rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At33"))>=0);
      void * d_At33 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At33_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::At33rhs"))>=0);
      void * d_At33rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_At33rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::B1"))>=0);
      void * d_B1 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_B1_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::B1rhs"))>=0);
      void * d_B1rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_B1rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::B2"))>=0);
      void * d_B2 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_B2_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::B2rhs"))>=0);
      void * d_B2rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_B2rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::B3"))>=0);
      void * d_B3 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_B3_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::B3rhs"))>=0);
      void * d_B3rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_B3rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::beta1"))>=0);
      void * d_beta1 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_beta1_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::beta1rhs"))>=0);
      void * d_beta1rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_beta1rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::beta2"))>=0);
      void * d_beta2 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_beta2_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::beta2rhs"))>=0);
      void * d_beta2rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_beta2rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::beta3"))>=0);
      void * d_beta3 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_beta3_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::beta3rhs"))>=0);
      void * d_beta3rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_beta3rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt11"))>=0);
      void * d_gt11 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt11_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt11rhs"))>=0);
      void * d_gt11rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt11rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt12"))>=0);
      void * d_gt12 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt12_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt12rhs"))>=0);
      void * d_gt12rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt12rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt13"))>=0);
      void * d_gt13 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt13_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt13rhs"))>=0);
      void * d_gt13rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt13rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt22"))>=0);
      void * d_gt22 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt22_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt22rhs"))>=0);
      void * d_gt22rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt22rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt23"))>=0);
      void * d_gt23 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt23_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt23rhs"))>=0);
      void * d_gt23rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt23rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt33"))>=0);
      void * d_gt33 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt33_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::gt33rhs"))>=0);
      void * d_gt33rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_gt33rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::phi"))>=0);
      void * d_phi = Device_GetVarI(cctkGH, vi, 0); 
      void * d_phi_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::phirhs"))>=0);
      void * d_phirhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_phirhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::trK"))>=0);
      void * d_trK = Device_GetVarI(cctkGH, vi, 0); 
      void * d_trK_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::trKrhs"))>=0);
      void * d_trKrhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_trKrhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::Xt1"))>=0);
      void * d_Xt1 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_Xt1_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::Xt1rhs"))>=0);
      void * d_Xt1rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_Xt1rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::Xt2"))>=0);
      void * d_Xt2 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_Xt2_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::Xt2rhs"))>=0);
      void * d_Xt2rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_Xt2rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::Xt3"))>=0);
      void * d_Xt3 = Device_GetVarI(cctkGH, vi, 0); 
      void * d_Xt3_out = Device_GetVarI(cctkGH, vi, 1);
    
      assert((vi = CCTK_VarIndex("ML_BSSN::Xt3rhs"))>=0);
      void * d_Xt3rhs = Device_GetVarI(cctkGH, vi, 0); 
      void * d_Xt3rhs_out = Device_GetVarI(cctkGH, vi, 1);
    
    
    size_t datasize = cctk_lsh[0] * cctk_lsh[1] * cctk_lsh[2];

    
    
    const int blocky = iDivUp(cctk_lsh[1] - stncl_yn - stncl_yp, CAKERNEL_Tiley - stncl_yn - stncl_yp);

    CaCUDA_Kernel_Launch_Parameters prms(cctk_iteration,
        cctk_lsh[0], cctk_lsh[1], cctk_lsh[2],
        cctk_nghostzones[0], cctk_nghostzones[1], cctk_nghostzones[2], blocky,
        cctk_delta_space[0], cctk_delta_space[1], cctk_delta_space[2],
        cctk_delta_time,
        cctk_origin_space[0], cctk_origin_space[1], cctk_origin_space[2],
        cctk_time);


    CAKERNEL_ML_BSSN_Advect<<<                                                    
 dim3(iDivUp(prms.cagh_ni - stncl_xn - stncl_xp, CAKERNEL_Tilex - stncl_xn - stncl_xp), 
      iDivUp(prms.cagh_nk - stncl_zn - stncl_zp, CAKERNEL_Tilez - stncl_zn - stncl_zp) 
          * blocky),
 dim3(CAKERNEL_Tilex, CAKERNEL_Threadsy, CAKERNEL_Threadsz)>>>(
      

(typeof(A)) d_A,(typeof(alpha)) d_alpha,(typeof(alpharhs)) d_alpharhs,(typeof(Arhs)) d_Arhs,(typeof(At11)) d_At11,(typeof(At11rhs)) d_At11rhs,(typeof(At12)) d_At12,(typeof(At12rhs)) d_At12rhs,(typeof(At13)) d_At13,(typeof(At13rhs)) d_At13rhs,(typeof(At22)) d_At22,(typeof(At22rhs)) d_At22rhs,(typeof(At23)) d_At23,(typeof(At23rhs)) d_At23rhs,(typeof(At33)) d_At33,(typeof(At33rhs)) d_At33rhs,(typeof(B1)) d_B1,(typeof(B1rhs)) d_B1rhs,(typeof(B2)) d_B2,(typeof(B2rhs)) d_B2rhs,(typeof(B3)) d_B3,(typeof(B3rhs)) d_B3rhs,(typeof(beta1)) d_beta1,(typeof(beta1rhs)) d_beta1rhs,(typeof(beta2)) d_beta2,(typeof(beta2rhs)) d_beta2rhs,(typeof(beta3)) d_beta3,(typeof(beta3rhs)) d_beta3rhs,(typeof(gt11)) d_gt11,(typeof(gt11rhs)) d_gt11rhs,(typeof(gt12)) d_gt12,(typeof(gt12rhs)) d_gt12rhs,(typeof(gt13)) d_gt13,(typeof(gt13rhs)) d_gt13rhs,(typeof(gt22)) d_gt22,(typeof(gt22rhs)) d_gt22rhs,(typeof(gt23)) d_gt23,(typeof(gt23rhs)) d_gt23rhs,(typeof(gt33)) d_gt33,(typeof(gt33rhs)) d_gt33rhs,(typeof(phi)) d_phi,(typeof(phirhs)) d_phirhs,(typeof(trK)) d_trK,(typeof(trKrhs)) d_trKrhs,(typeof(Xt1)) d_Xt1,(typeof(Xt1rhs)) d_Xt1rhs,(typeof(Xt2)) d_Xt2,(typeof(Xt2rhs)) d_Xt2rhs,(typeof(Xt3)) d_Xt3,(typeof(Xt3rhs)) d_Xt3rhs,



 d_pars.ptr, 
d_pars.LapseAdvectionCoeff_offset, d_pars.ShiftBCoeff_offset, d_pars.LapseACoeff_offset, d_pars.ShiftAdvectionCoeff_offset, 
 prms);
//    cutilCheckMsg("failed while updating the velocity");
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    
    
}
